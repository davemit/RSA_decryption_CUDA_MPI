#include "hip/hip_runtime.h"
#include "pollardCuda.h"



int Isprime (long long int);


long long int prime(long long int num,long long int** primearray)

{
    long long int   count, c;
    long long int i=3;
    long long int *tmp = (long long int*) malloc(num*(sizeof(long long int)));
    
    for ( count = 2 ; count <= num ;  )
    {
        for ( c = 2 ; c <= i - 1 ; c++ ) {
        if ( i%c == 0 )
        break;
        }
        if ( c == i )
        {
           
            tmp[0]=2;
            tmp[count-1]=i;

            *primearray=tmp;
            
            count++;
        }
        i++;
    }
        //printf("abcd %ld",tmp[8]);
    return tmp[count-2];
}

/*****************************************
 * 
 * 
 *Checking if the number is prime number 
 * 
 * 
 * 
 * 
 * 
 *****************************************/
int IsPrime(long long int num)
{
	long long int j;
	long long int k;
	k = sqrt(num);
	
	for (j=2;j<=k;j++)
	{
		//printf("\nHere");
		if(num%j==0)
		return 0;
	}
	return 1;
}



int main()
{
	long long int *d_primearray;
	long long int *h_primearray;
	long long int bound=75;
	long long int h_highestPrime; 
	long long int *res;
	
	
	
	
	long long int other_factor,after_e;
	long long int p, q, n;
	int flag;
	
	long long int t;
	
	
	printf("Enter the prime number\n");
	scanf("%lld",&p);
	
	flag = IsPrime(p);
	if( flag == 0)
	{
		printf("wrong input");
		exit(0);
	}
	printf("Enter another prime number\n");
	scanf("%lld",&q);
	flag = IsPrime(q);
	if(flag ==0 || p==q)
	{
		printf("wrong input");
		exit(0);
	}
	
	
	n= p*q;
	t = (p-1)*(q-1);
	
	
	/*************************************************/
	
	h_highestPrime = prime(bound, &h_primearray);
	
 	hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipMalloc((void**)&d_primearray,bound*sizeof(long long int));
    hipMalloc((void**)&res,sizeof(long long int));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_cmalloc = 0;
    hipEventElapsedTime(&time_cmalloc,start,stop);
	hipEventRecord(start);
    hipMemcpy(d_primearray,h_primearray,bound*sizeof(long long int),hipMemcpyHostToDevice);
	hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_htod=0;
	double NUM_BLOCKS;
	NUM_BLOCKS = ceil((double)bound /NUM_THREADS);
	
    hipEventElapsedTime(&time_htod,start,stop);
    
	hipEventRecord(start);
	
    pollard_gpu<<<(int)NUM_BLOCKS,NUM_THREADS>>>(d_primearray,h_highestPrime,bound,res, n);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    
    float time_kernel = 0;
    hipEventElapsedTime(&time_kernel,start,stop);
	long long int *gpu_res;
	gpu_res = (long long int*) malloc(sizeof(long long int));
	hipEventRecord(start);
    hipMemcpy(gpu_res,res,sizeof(long long int),hipMemcpyDeviceToHost);
	printf("Factor = %lld\n",*gpu_res);
	
	
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_dtoh = 0;
    hipEventElapsedTime(&time_dtoh,start,stop);
	hipFree(d_primearray);
	hipFree(res);
	other_factor = n/ (*gpu_res);
	after_e = ((*gpu_res)-1)*(other_factor-1);
	
	
  	printf("Device malloc = %f ms\n",time_cmalloc);
    printf("Device to Host = %f ms\n",time_htod);
    printf("Kernel execution = %f ms\n",time_kernel);        
	printf("Host to Device = %f ms\n",time_dtoh);
    return 0;                                          
}

